#include "CudaTimer.cuh"

CudaTimer::CudaTimer() {
    hipEventCreate(&startTime);
    hipEventCreate(&stopTime);
    elapsedTimeMiliseconds = 0.0f;
}

CudaTimer::~CudaTimer() {
    hipEventDestroy(startTime);
    hipEventDestroy(stopTime);
}

void CudaTimer::start() {
    hipEventRecord(startTime);
}

void CudaTimer::stop() {
    hipEventRecord(stopTime);
    hipEventSynchronize(stopTime);
    hipEventElapsedTime(&elapsedTimeMiliseconds, startTime, stopTime);
}

float CudaTimer::getElapsedTimeMiliseconds() const {
    return elapsedTimeMiliseconds;
}
