#include "hip/hip_runtime.h"
#include "CellularAutomata/CADataPrinter.cuh"
#include "CellularAutomata/Solvers/AMXSolver.cuh"
#include "Memory/Allocators/CPUAllocator.cuh"

AMXSolver::AMXSolver(CADataDomain<uint8_t>* domain, CADataDomain<uint8_t>* domainBuffer) {
    dataDomain = domain;
    dataDomainBuffer = domainBuffer;

    CPUAllocator<int>* cpuAllocator = new CPUAllocator<int>();
    Allocator<int>* allocator = reinterpret_cast<Allocator<int>*>(cpuAllocator);
    hostVisibleData = new CADataDomain<int>(allocator, dataDomain->getInnerHorizontalSize(), dataDomain->getHorizontalHaloSize());
    hostVisibleData->allocate();

    setupAMX();
    
    fillTridiag();
}

void AMXSolver::setupAMX() {
    __tilecfg tile_config;

    tile_config->palette_id = 1;
    tile_config->start_row = 0;

    // Configure tiles for block_size x block_size matrices
    for (int i = 0; i < 1; ++i) {
        tile_config->colsb[i] = 16 ;
        tile_config->rows[i] = 16;
    }
    for (int i = 1; i < 8; ++i) {
        tile_config->colsb[i] = 64;
        tile_config->rows[i] = 16;
    }

    _tile_loadconfig(tile_config);
}


void AMXSolver::fillTridiag() {
    int i;

    for (i = 0; i < 16*64; i += 1)
    {
        int col = i & 15;
        int row = i >> 4;
        if (col + 15 - RADIUS < row){

            pi_1[i] = 1;
        } else {
            pi_1[i] = 0;
        }
    }

    for (i = 0; i < 16*64; i += 1)
    {
        int col = i & 15;
        int row = i >> 4;
        if (abs(col - row) <= RADIUS){
            pi_2[i] = 1;
        } else {
            pi_2[i] = 0;
        }
    }
    for (i = 0; i < 16*64; i += 1)
    {
        int col = i & 15;
        int row = i >> 4;
        if (col -15+RADIUS> row){
            pi_3[i] = 1;
        } else {
            pi_3[i] = 0;
        }
    }

    // // debug print tridiag in 2d
    // for (int i = 0; i < 16; i++) {
    //     for (int j = 0; j < 16; j++) {
    //         std::cout << (int)tridiag[i * 16 + j] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // for (int i = 0; i < 16; i++) {
    //     for (int j = 0; j < 16; j++) {
    //         std::cout << (int)tridiag[i * 16 + j + 256] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // for (int i = 0; i < 16; i++) {
    //     for (int j = 0; j < 16; j++) {
    //         std::cout << (int)tridiag[i * 16 + j + 512] << " ";
    //     }
    //     std::cout << std::endl;
    // }


}

void AMXSolver::copyCurrentStateToHostVisibleData() {
    for (int i = 0; i < dataDomain->getTotalSize(); ++i) {
        uint8_t value = dataDomain->getElementAt(i);
        hostVisibleData->setElementAt(i, (int)value);
    }
}
void AMXSolver::copyHostVisibleDataToCurrentState() {
    for (int i = 0; i < hostVisibleData->getTotalSize(); ++i) {
        int value = hostVisibleData->getElementAt(i);
        dataDomain->setElementAt(i, value);
    }
}

void AMXSolver::swapPointers() {
    CADataDomain<uint8_t>* temp = dataDomain;
    dataDomain = dataDomainBuffer;
    dataDomainBuffer = temp;
}

uint8_t AMXSolver::transitionFunction(int k, int a, int b) {
    return (1 - (((k - a) >> 31) & 0x1)) * (1 - (((b - k) >> 31) & 0x1));
}

void AMXSolver::CAStepAlgorithm() {
    uint8_t* data = dataDomain->getData();
    size_t nWithHalo = dataDomain->getFullHorizontalSize();

    _tile_loadd(1, pi_1, 64);
    _tile_loadd(2, pi_2, 64);
    _tile_loadd(3, pi_3, 64);

    //FIRST STEP: horizontal reduction
    for (int i = 0; i < nWithHalo; i+=16) {
        for (int j = 0; j < nWithHalo - 16*2; j+=16) {
            //take three continuous 16x16 blocks and load them into amx
            _tile_zero(0);
            

            _tile_loadd(4, data + i * nWithHalo + j, nWithHalo);
            _tile_loadd(5, data + i * nWithHalo + j + 16, nWithHalo);
            _tile_loadd(6, data + i * nWithHalo + j + 32, nWithHalo);

            _tile_dpbssd(0, 4, 1);
            _tile_dpbssd(0, 5, 2);
            _tile_dpbssd(0, 6, 3);

            _tile_stored(0, data + i * dataDomainBuffer->getFullHorizontalSize() + j+16, dataDomainBuffer->getFullHorizontalSize());
        }
    }


    // for (int i = 0; i < dataDomain->getInnerHorizontalSize(); ++i) {
    //     for (int j = 0; j < dataDomain->getInnerHorizontalSize(); ++j) {
    //         int liveNeighbors = countAliveNeighbors(i, j);
    //         uint8_t cellValue = dataDomain->getInnerElementAt(i, j);
    //         uint8_t result = cellValue * transitionFunction(liveNeighbors, SMIN, SMAX) + (1 - cellValue) * transitionFunction(liveNeighbors, BMIN, BMAX);

    //         dataDomainBuffer->setInnerElementAt(i, j, result);
    //     }
    // }
}

int AMXSolver::countAliveNeighbors(int y, int x) {
    int aliveNeighbors = 0;

    for (int i = -RADIUS; i <= RADIUS; ++i) {
        for (int j = -RADIUS; j <= RADIUS; ++j) {
            if (i == 0 && j == 0)
                continue;
            aliveNeighbors += dataDomain->getInnerElementAt(y + i, x + j);
        }
    }

    return aliveNeighbors;
}

void AMXSolver::fillHorizontalBoundaryConditions() {
    for (int h = 0; h < dataDomain->getHorizontalHaloSize(); ++h) {
        for (int j = 0; j < dataDomain->getInnerHorizontalSize(); ++j) {
            size_t topIndex = (dataDomain->getHorizontalHaloSize() + h) * dataDomain->getFullHorizontalSize() + dataDomain->getHorizontalHaloSize() + j;
            size_t bottomIndex = topIndex + (dataDomain->getInnerHorizontalSize()) * dataDomain->getFullHorizontalSize();
            uint8_t value = dataDomain->getElementAt(topIndex);
            dataDomain->setElementAt(bottomIndex, value);
        }

        for (int j = 0; j < dataDomain->getInnerHorizontalSize(); ++j) {
            size_t topIndex = (h)*dataDomain->getFullHorizontalSize() + dataDomain->getHorizontalHaloSize() + j;
            size_t bottomIndex = topIndex + (dataDomain->getInnerHorizontalSize()) * dataDomain->getFullHorizontalSize();

            uint8_t value = dataDomain->getElementAt(bottomIndex);
            dataDomain->setElementAt(topIndex, value);
        }
    }
}

void AMXSolver::fillVerticalBoundaryConditions() {
    for (int h = 0; h < dataDomain->getHorizontalHaloSize(); ++h) {
        for (int i = 0; i < dataDomain->getFullHorizontalSize(); ++i) {
            size_t leftIndex = i * dataDomain->getFullHorizontalSize() + h;
            size_t rightIndex = leftIndex + dataDomain->getInnerHorizontalSize();
            uint8_t value = dataDomain->getElementAt(rightIndex);
            dataDomain->setElementAt(leftIndex, value);
        }

        for (int i = 0; i < dataDomain->getFullHorizontalSize(); ++i) {
            size_t leftIndex = i * dataDomain->getFullHorizontalSize() + dataDomain->getHorizontalHaloSize() + h;
            size_t rightIndex = leftIndex + dataDomain->getInnerHorizontalSize();
            uint8_t value = dataDomain->getElementAt(leftIndex);
            dataDomain->setElementAt(rightIndex, value);
        }
    }
}
