#include "hip/hip_runtime.h"
#include "StatsCollector.hpp"
#include <cinttypes>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>

#include "TensorCA2D.cuh"

#define PRINT_LIMIT 7

// change to runtime parameter
const uint32_t STEPS = 10;

int main(int argc, char** argv) {
    // srand ( time(NULL) );
    if (argc != 7) {
        printf("run as ./prog <deviceId> <n> <mode> <repeats> <density> <seed>\n");
        exit(1);
    }

    debugInit(5, "log.txt");
    uint32_t deviceId = atoi(argv[1]);
    uint32_t n = atoi(argv[2]);
    uint32_t mode = atoi(argv[3]);
    uint32_t repeats = atoi(argv[4]);
    float density = atof(argv[5]);
    uint32_t seed = atoi(argv[6]);

    StatsCollector stats;
    TensorCA2D* benchmark;

    for (int i = 0; i < repeats; i++) {
        benchmark = new TensorCA2D(deviceId, n, mode, density);
        if (!benchmark->init(seed)) {
            exit(1);
        }
        float iterationTime = benchmark->doBenchmarkAction(STEPS);
        // benchmark->transferDeviceToHost();
        stats.add(iterationTime);
        if (i != repeats - 1) {
            delete benchmark;
        }
    }

    benchmark->transferDeviceToHost();
    fDebug(1, benchmark->printHostData());

#ifdef VERIFY
    TensorCA2D* reference = new TensorCA2D(deviceId, n, 0, density);
    if (!reference->init(seed)) {
        exit(1);
    }
    reference->doBenchmarkAction(STEPS);
    reference->transferDeviceToHost();
    fDebug(1, reference->printHostData());

    printf("main(): avg kernel time: %f ms\n", stats.getAverage());
    printf("\x1b[0m");
    fflush(stdout);
    if (!reference->compare(benchmark)) {
        printf("\n[VERIFY] verification FAILED!.\n\n");

        exit(1);
    }

    printf("\n[VERIFY] verification successful.\n\n");

#endif

#ifdef DEBUG
    printf("maxlong %lu\n", LONG_MAX);
    printf("\x1b[1m");
    fflush(stdout);
    printf("main(): avg kernel time: %f ms\n", stats.getAverage());
    printf("\x1b[0m");
    fflush(stdout);
#else
    printf("%f, %f, %f, %f\n", stats.getAverage(), stats.getStandardDeviation(), stats.getStandardError(), stats.getVariance());
#endif
}
