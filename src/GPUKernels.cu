#include "hip/hip_runtime.h"
#ifndef _CLASSIC_GOL_KERNELS_H_
#define _CLASSIC_GOL_KERNELS_H_
#include "GPUKernels.cuh"

#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>

using namespace nvcuda;

#define CELL_NEIGHBOURS 8

__device__ inline int h(int k, int a, int b)
{
    return (1 - (((k - a) >> 31) & 0x1)) * (1 - (((b - k) >> 31) & 0x1));
}

__forceinline__ __device__ void workWithShmem(char *pDataOut, char *shmem, uint2 dataCoord, uint32_t nWithHalo,
                                              uint32_t nShmem)
{
    int nc = 0;
#pragma unroll
    for (int i = -RADIUS; i <= RADIUS; i++)
    {
        for (int j = -RADIUS; j <= RADIUS; j++)
        {
            nc += shmem[HINDEX(threadIdx.x + j, threadIdx.y + i, nShmem)];
        }
    }
    unsigned int c = shmem[HINDEX(threadIdx.x, threadIdx.y, nShmem)];
    nc -= c;
    pDataOut[HINDEX(dataCoord.x, dataCoord.y, nWithHalo)] = c * h(nc, SMIN, SMAX) + (1 - c) * h(nc, BMIN, BMAX);
}

__forceinline__ __device__ void workWithGbmem(char *pDataIn, char *pDataOut, uint2 dataCoord, uint32_t nWithHalo)
{
    int nc = 0;
#pragma unroll
    for (int i = -RADIUS; i <= RADIUS; i++)
    {
        for (int j = -RADIUS; j <= RADIUS; j++)
        {
            nc += pDataIn[HINDEX(dataCoord.x + j, dataCoord.y + i, nWithHalo)];
        }
    }
    unsigned int c = pDataIn[HINDEX(dataCoord.x, dataCoord.y, nWithHalo)];
    nc -= c;
    pDataOut[HINDEX(dataCoord.x, dataCoord.y, nWithHalo)] = c * h(nc, SMIN, SMAX) + (1 - c) * h(nc, BMIN, BMAX);
}

__global__ void BASE_KERNEL(char *pDataIn, char *pDataOut, size_t n, size_t nWithHalo)
{
    uint32_t dataBlockCoord_x = blockIdx.x * blockDim.x;
    uint32_t dataBlockCoord_y = blockIdx.y * blockDim.y;
    uint2 dataCoord = {dataBlockCoord_x + threadIdx.x, dataBlockCoord_y + threadIdx.y};
    if (dataCoord.x < n && dataCoord.y < n)
    {
        workWithGbmem(pDataIn, pDataOut, dataCoord, nWithHalo);
    }
}

__global__ void COARSE_KERNEL(char *pDataIn, char *pDataOut, size_t n, size_t nWithHalo)
{
    __shared__ char shmem[(BMAXLLSHMEM_N) * (BMAXLLSHMEM_N)];
    uint32_t tid = threadIdx.y * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.y * gridDim.x + blockIdx.x;
    uint32_t dataBlockCoord_x = blockIdx.x * 80;
    uint32_t dataBlockCoord_y = blockIdx.y * 80;

    for (uint32_t i = tid; i < BMAXLLSHMEM_N * BMAXLLSHMEM_N; i += BSIZE3DX * BSIZE3DY)
    {
        uint32_t shmem_x = i % BMAXLLSHMEM_N;
        uint32_t shmem_y = i / BMAXLLSHMEM_N;
        uint32_t data_x = dataBlockCoord_x + shmem_x;
        uint32_t data_y = dataBlockCoord_y + shmem_y;
        if (data_x < nWithHalo && data_y < nWithHalo)
        {
            shmem[GINDEX(shmem_x, shmem_y, BMAXLLSHMEM_N)] = pDataIn[GINDEX(data_x, data_y, nWithHalo)];
        }
    }
    __syncthreads();
    for (uint32_t i = tid; i < 80 * 80; i += BSIZE3DX * BSIZE3DY)
    {
        uint32_t shmem_x = i % 80;
        uint32_t shmem_y = i / 80;
        uint32_t data_x = dataBlockCoord_x + shmem_x;
        uint32_t data_y = dataBlockCoord_y + shmem_y;
        uint2 dataCoord = {data_x, data_y};
        if (dataCoord.x < n && dataCoord.y < n)
        {
            int nc = 0;
            for (int i = -RADIUS; i <= RADIUS; i++)
            {
                for (int j = -RADIUS; j <= RADIUS; j++)
                {
                    nc += shmem[HINDEX(shmem_x + j, shmem_y + i, BMAXLLSHMEM_N)];
                }
            }
            unsigned int c = shmem[HINDEX(shmem_x, shmem_y, BMAXLLSHMEM_N)];
            nc -= c;
            pDataOut[HINDEX(dataCoord.x, dataCoord.y, nWithHalo)] = c * h(nc, SMIN, SMAX) + (1 - c) * h(nc, BMIN, BMAX);
        }
    }
}

__global__ void CAT_KERNEL(half *pDataIn, half *pDataOut, size_t n, size_t nWithHalo)
{
    const uint32_t nFragmentsH = NREGIONS_H + 2;

    extern __shared__ char totalshmem[];
    half *shmem = (half *)totalshmem;

    __shared__ half shmem_tridiag[16 * 16 * 2];

    const uint32_t tid = threadIdx.y * blockDim.x + threadIdx.x;
    const uint32_t wid = tid / 32;

    int i;
#pragma unroll
    for (i = tid; i < 256; i += BSIZE3DX * BSIZE3DY)
    {
        //  printf("%u,%u = %.0f\n", i, index, __half2float(tridiagTemplate[index]));
        shmem_tridiag[i] = (17 + RADIUS - abs((i >> 4) - (i & 15))) / 17; // tridiagTemplate[index];
    }
#pragma unroll
    for (i = tid; i < 256; i += BSIZE3DX * BSIZE3DY)
    {
        shmem_tridiag[i + 16 * 16] =
            (16 - (i & 15) + (i >> 4)) / (32 - RADIUS); //(((i >> 4) + 1) >> 4) * ((16 - (i & 15)) >> 4);
    }

    __syncthreads();

    wmma::fragment<wmma::accumulator, 16, 16, 16, half> c_frag;
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag2;
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag3;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fill_fragment(c_frag, 0);

    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> T_0_asB; // Row major
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> T_1_asB; // Row major
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> T_2_asB; // Col major

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> T_0_asA; // Col major
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> T_1_asA; // Row major
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> T_2_asA; // Row major

    const uint8_t wcount = (BSIZE3DX * BSIZE3DY) / 32;

    const uint32_t n16 = n >> 4;
    const uint32_t nWithHalo16 = nWithHalo >> 4;
#pragma unroll

    for (uint32_t rid = wid; rid < NREGIONS_H * (NREGIONS_V + 2); rid += wcount)
    {
        const uint32_t workFragment_x = (rid % NREGIONS_H);
        const uint32_t workFragment_y = (rid / NREGIONS_H);
        const uint32_t regionCoord_x = blockIdx.x * NREGIONS_H;
        const uint32_t regionCoord_y = blockIdx.y * NREGIONS_V;
        // for (char fragRow = 0; i < 8; i += 1) {
        const uint32_t globalFragment_x = regionCoord_x + workFragment_x;
        const uint32_t globalFragment_y = regionCoord_y + workFragment_y;

        if (!(globalFragment_x < n16 && globalFragment_y < nWithHalo16))
        {
            continue;
        }

        size_t globalFragment_p = (globalFragment_y * nWithHalo16 + globalFragment_x) << 8;

        wmma::load_matrix_sync(a_frag, &pDataIn[globalFragment_p], 16);
        wmma::load_matrix_sync(a_frag2, &pDataIn[globalFragment_p + 256], 16);
        wmma::load_matrix_sync(a_frag3, &pDataIn[globalFragment_p + 512], 16);

        wmma::load_matrix_sync(T_0_asB, &shmem_tridiag[256], 16);
        wmma::load_matrix_sync(T_2_asB, &shmem_tridiag[256], 16);
        wmma::load_matrix_sync(T_1_asB, shmem_tridiag, 16);

        wmma::mma_sync(c_frag, a_frag, T_0_asB, c_frag);
        wmma::mma_sync(c_frag, a_frag2, T_1_asB, c_frag);
        wmma::mma_sync(c_frag, a_frag3, T_2_asB, c_frag);

        wmma::store_matrix_sync(&shmem[workFragment_y * nFragmentsH * 256 + (workFragment_x + 1) * 256], c_frag, 16,
                                wmma::mem_row_major);
        wmma::fill_fragment(c_frag, 0.0f);
    }

    __syncthreads();
#pragma unroll

    for (uint32_t rid = wid; rid < NREGIONS_H * (NREGIONS_V); rid += wcount)
    {
        const uint32_t workFragment_x = rid % NREGIONS_H;
        const uint32_t workFragment_y = rid / NREGIONS_H;
        const uint32_t regionCoord_x = blockIdx.x * NREGIONS_H;
        const uint32_t regionCoord_y = blockIdx.y * NREGIONS_V;

        uint32_t globalFragment_x = regionCoord_x + workFragment_x;
        uint32_t globalFragment_y = regionCoord_y + workFragment_y;

        if (globalFragment_x >= n16 || globalFragment_y >= n16)
        {
            continue;
        }
        size_t globalFragment_p = (workFragment_y * nFragmentsH + (workFragment_x + 1)) * 256;
        wmma::load_matrix_sync(b_frag, &shmem[globalFragment_p], 16);
        wmma::load_matrix_sync(T_0_asA, &shmem_tridiag[256], 16);
        wmma::mma_sync(c_frag, T_0_asA, b_frag, c_frag);

        wmma::load_matrix_sync(b_frag, &shmem[globalFragment_p + nFragmentsH * 256], 16);
        wmma::load_matrix_sync(T_1_asA, shmem_tridiag, 16);
        wmma::mma_sync(c_frag, T_1_asA, b_frag, c_frag);

        wmma::load_matrix_sync(b_frag, &shmem[globalFragment_p + nFragmentsH * 512], 16);
        wmma::load_matrix_sync(T_2_asA, &shmem_tridiag[256], 16);
        wmma::mma_sync(c_frag, T_2_asA, b_frag, c_frag);

        wmma::store_matrix_sync(&pDataOut[((globalFragment_y + 1) * nWithHalo16 + (globalFragment_x + 1)) * 256],
                                c_frag, 16, wmma::mem_row_major);
        wmma::fill_fragment(c_frag, 0.0f);
    }

    __syncthreads();
#pragma unroll

    for (uint32_t index = tid; index < NREGIONS_H * 16 * NREGIONS_V * 16; index += BSIZE3DX * BSIZE3DY)
    {
        uint32_t fid = index >> 8;
        uint32_t fx = fid % NREGIONS_H;
        uint32_t fy = fid / NREGIONS_H;

        uint32_t regionCoord_x = (blockIdx.x) * NREGIONS_H;
        uint32_t regionCoord_y = (blockIdx.y) * NREGIONS_V;

        uint32_t globalFragment_x = regionCoord_x + fx + 1;
        uint32_t globalFragment_y = regionCoord_y + fy + 1;

        size_t dindex = (globalFragment_y * nWithHalo16 + globalFragment_x) * 256 + (index & 255);
        if (globalFragment_x < (nWithHalo16)-1 && globalFragment_y < (nWithHalo16)-1)
        {
            uint32_t val = __half2uint_rn(pDataOut[dindex]);
            float val2 = __half2float(pDataIn[dindex]);
            pDataOut[dindex] =
                __uint2half_rn(val2 * h(val - val2, SMIN, SMAX) + (1 - val2) * h(val - val2, BMIN, BMAX));
        }
    }
}

__global__ void convertFp32ToFp16(half *out, int *in, int nWithHalo)
{
    int tx = blockDim.x * blockIdx.x + threadIdx.x;
    int ty = blockDim.y * blockIdx.y + threadIdx.y;
    if (tx < nWithHalo && ty < nWithHalo)
    {
        out[tx + ty * (size_t)nWithHalo] = __uint2half_rn(in[tx + ty * (size_t)nWithHalo]);
    }
}
__global__ void convertFp16ToFp32(int *out, half *in, int nWithHalo)
{
    int tx = blockDim.x * blockIdx.x + threadIdx.x;
    int ty = blockDim.y * blockIdx.y + threadIdx.y;
    if (tx < nWithHalo && ty < nWithHalo)
    {
        out[tx + ty * (size_t)nWithHalo] = __half2uint_rn(in[tx + ty * (size_t)nWithHalo]);
    }
}

__global__ void convertFp32ToFp16AndDoChangeLayout(half *out, int *in, size_t nWithHalo)
{
    uint32_t tx = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t ty = blockDim.y * blockIdx.y + threadIdx.y;
    size_t tid = threadIdx.y * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.y * gridDim.x + blockIdx.x;

    if (tx < nWithHalo && ty < nWithHalo)
    {
        out[bid * 256 + tid] = __uint2half_rd(in[ty * nWithHalo + tx]);
    }
}
__global__ void convertFp16ToFp32AndUndoChangeLayout(int *out, half *in, size_t nWithHalo)
{
    uint32_t tx = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t ty = blockDim.y * blockIdx.y + threadIdx.y;
    size_t tid = threadIdx.y * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.y * gridDim.x + blockIdx.x;

    if (tx < nWithHalo && ty < nWithHalo)
    {
        out[ty * nWithHalo + tx] = __half2uint_rn(in[bid * 256 + tid]);
    }
}

__global__ void convertUInt32ToUInt4AndDoChangeLayout(int *out, char *in, size_t nWithHalo)
{
    size_t tid = threadIdx.y * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.y * gridDim.x + blockIdx.x;
    uint32_t tx = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t ty = blockDim.y * blockIdx.y + threadIdx.y;

    if (tx < nWithHalo && ty < nWithHalo)
    {
        int val = 0;
#pragma unroll
        for (int i = 0; i < 8; i++)
        {
            val |= (in[ty * nWithHalo + (tx) * 8 + i] & 0b1111) << (i * 4);
        }
        out[bid * 1024 / 8 + tid] = val;
    }
}
__global__ void convertUInt4ToUInt32AndUndoChangeLayout(char *out, int *in, size_t nWithHalo)
{
    size_t tid = threadIdx.y * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.y * gridDim.x + blockIdx.x;
    uint32_t tx = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t ty = blockDim.y * blockIdx.y + threadIdx.y;

    if (tx < nWithHalo && ty < nWithHalo)
    {
        int val = in[(bid * 1024 / 8 + tid)];
#pragma unroll
        for (int i = 0; i < 8; i++)
        {
            out[ty * nWithHalo + (tx) * 8 + i] = (val >> (i * 4)) & 0b1111;
        }
    }
}
__global__ void UndoChangeLayout(char *out, char *in, size_t nWithHalo)
{
    uint32_t tx = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t ty = blockDim.y * blockIdx.y + threadIdx.y;
    size_t tid = threadIdx.y * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.y * gridDim.x + blockIdx.x;

    // printf("%i, %i -> %i, %i\n", tx, ty, in_x, in_y);
    // printf("%llu -> %llu\n", tx + ty * nWithHalo, bid*256+tid);

    if (tx < nWithHalo && ty < nWithHalo)
    {
        out[ty * nWithHalo + tx] = in[bid * 1024 + tid];
    }
}

__global__ void onlyConvertUInt32ToUInt4(int *out, char *in, size_t nWithHalo)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < nWithHalo * nWithHalo / 8)
    {
        int val = 0;
#pragma unroll
        for (int i = 0; i < 8; i++)
        {
            val |= (in[tid * 8 + i] & 0b1111) << (i * 4);
        }
        out[tid] = val;
    }
}

__global__ void convertInt32ToInt8AndDoChangeLayout(unsigned char *out, int *in, size_t nWithHalo)
{
    uint32_t tx = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t ty = blockDim.y * blockIdx.y + threadIdx.y;
    size_t tid = threadIdx.y * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.y * gridDim.x + blockIdx.x;

    if (tx < nWithHalo && ty < nWithHalo)
    {
        out[bid * 256 + tid] = (unsigned char)(in[ty * nWithHalo + tx]);
    }
}
__global__ void convertInt8ToInt32AndUndoChangeLayout(int *out, unsigned char *in, size_t nWithHalo)
{
    uint32_t tx = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t ty = blockDim.y * blockIdx.y + threadIdx.y;
    size_t tid = threadIdx.y * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.y * gridDim.x + blockIdx.x;

    if (tx < nWithHalo && ty < nWithHalo)
    {
        out[ty * nWithHalo + tx] = (int)(in[bid * 256 + tid]);
    }
}

__global__ void copyHorizontalHalo(char *data, size_t n, size_t nWithHalo)
{
    // We want id ∈ [1,dim]
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id < n)
    {
#pragma unroll

        for (int i = 0; i < RADIUS; i++)
        {
            // Copy first real row to bottom ghost row
            data[(nWithHalo * (n + RADIUS + i)) + (id + RADIUS)] = data[(nWithHalo * (RADIUS + i)) + id + RADIUS];
            // Copy last real row to top ghost row
            data[nWithHalo * i + id + RADIUS] = data[(nWithHalo) * (n + i) + id + RADIUS];
        }
    }
}

__global__ void copyVerticalHalo(char *data, size_t n, size_t nWithHalo)
{
    // We want id ∈ [0,dim+1]
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id < nWithHalo)
    {
#pragma unroll
        for (int i = 0; i < RADIUS; i++)
        {
            // Copy first real column to right most ghost column
            data[(id) * (nWithHalo) + (n + RADIUS + i)] = data[(id) * (nWithHalo) + (RADIUS + i)];
            // Copy last real column to left most ghost column
            data[(id) * (nWithHalo) + i] = data[(id) * (nWithHalo) + (n + i)];
        }
    }
}

__global__ void copyHorizontalHaloCoalescedVersion(half *data, size_t n, size_t nWithHalo)
{
    size_t tid = threadIdx.y * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.y * gridDim.x + blockIdx.x;

    if (bid < n / 16)
    {
        data[(bid + 1) * 256 + tid] = data[(bid + 1 + nWithHalo / 16 * n / 16) * 256 + tid];
    }
    else if (bid < 2 * (n / 16))
    {
        bid -= n / 16;
        data[(bid + 1 + nWithHalo / 16 * (nWithHalo / 16 - 1)) * 256 + tid] =
            data[(bid + 1 + nWithHalo / 16) * 256 + tid];
    }
}

__global__ void copyVerticalHaloCoalescedVersion(half *data, size_t n, size_t nWithHalo)
{
    size_t tid = threadIdx.y * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.y * gridDim.x + blockIdx.x;

    if (bid < nWithHalo / 16)
    {
        data[(bid * (nWithHalo / 16) * 256) + tid] = data[(bid * (nWithHalo / 16) * 256) + (n / 16) * 256 + tid];
    }
    else if (bid < 2 * (nWithHalo / 16))
    {
        bid -= nWithHalo / 16;
        // printf("ASD\n");
        data[(bid * (nWithHalo / 16) * 256) + (n / 16 + 1) * 256 + tid] =
            data[(bid * (nWithHalo / 16) * 256) + tid + 256];
    }
}
__global__ void copyHorizontalHaloHalf(half *data, size_t n, size_t nWithHalo)
{
    // We want id ∈ [1,dim]
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id < n)
    {
#pragma unroll

        for (int i = 0; i < RADIUS; i++)
        {
            // Copy first real row to bottom ghost row
            data[(nWithHalo * (n + RADIUS + i)) + (id + RADIUS)] = data[(nWithHalo * (RADIUS + i)) + id + RADIUS];
            // Copy last real row to top ghost row
            data[nWithHalo * i + id + RADIUS] = data[(nWithHalo) * (n + i) + id + RADIUS];
        }
    }
}

__global__ void copyVerticalHaloHalf(half *data, size_t n, size_t nWithHalo)
{
    // We want id ∈ [0,dim+1]
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id < nWithHalo)
    {
#pragma unroll
        for (int i = 0; i < RADIUS; i++)
        {
            // Copy first real column to right most ghost column
            data[(id) * (nWithHalo) + (n + RADIUS + i)] = data[(id) * (nWithHalo) + (RADIUS + i)];
            // Copy last real column to left most ghost column
            data[(id) * (nWithHalo) + i] = data[(id) * (nWithHalo) + (n + i)];
        }
    }
}

__global__ void copyHorizontalHaloTensor(half *data, size_t n, size_t nWithHalo)
{
    // We want id ∈ [1,dim]
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (j < n)
    {
#pragma unroll

        for (int h = 0; h < RADIUS; h++)
        {
            // Copy last real row to top ghost row
            data[(nWithHalo * (h + 16 - RADIUS)) + j + 16] = data[(nWithHalo) * (n + (h + 16 - RADIUS)) + j + 16];
            // Copy first real row to bottom ghost row
            data[(nWithHalo * (n + h + 16)) + (j + 16)] = data[(nWithHalo * (16 + h)) + j + 16];
        }
    }
}

__global__ void copyVerticalHaloTensor(half *data, size_t n, size_t nWithHalo)
{
    // We want id ∈ [0,dim+1]
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < nWithHalo)
    {
#pragma unroll
        for (int h = 0; h < RADIUS; h++)
        {
            // Copy first real column to right most ghost column
            data[(i) * (nWithHalo) + (n + 16 + h)] = data[(i) * (nWithHalo) + (16 + h)];
            // Copy last real column to left most ghost column
            data[(i) * (nWithHalo) + (h + 16 - RADIUS)] = data[(i) * (nWithHalo) + (n + (h + 16 - RADIUS))];
        }
    }
}

__global__ void copyFromMTYPEAndCast(char *from, int *to, size_t nWithHalo)
{
    size_t tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t tid = tid_y * blockDim.x * gridDim.x + tid_x;
    for (size_t index = tid; index < nWithHalo * nWithHalo; index += blockDim.x * blockDim.y * gridDim.x * gridDim.y)
    {
        to[index] = (int)from[index];
    }
}
__global__ void copyToMTYPEAndCast(int *from, char *to, size_t nWithHalo)
{
    size_t tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t tid = tid_y * blockDim.x * gridDim.x + tid_x;
    for (size_t index = tid; index < nWithHalo * nWithHalo; index += blockDim.x * blockDim.y * gridDim.x * gridDim.y)
    {
        to[index] = (char)from[index];
    }
}

///////////////////////////////////////////////////////////
#define sh_row (size_t) threadIdx.y
#define sh_col ((size_t)threadIdx.x * cellsPerThread)
#define x2 ((size_t)x * cellsPerThread)
#define sh_size_x (blockDim.x * cellsPerThread)
__forceinline__ __device__ int count_neighs(char c, int my_id, int size_i, char *lattice, int neighs, int halo);

__global__ void MCELL_KERNEL(char *d_lattice, char *d_lattice_new, int size_i, int size_j, int cellsPerThread,
                             int neighs, int halo)
{

    const size_t totalShmem = ((BSIZE3DX * 2 + 2 * RADIUS) * (BSIZE3DY + 2 * RADIUS));
    const size_t sh_stride = ((BSIZE3DX * 2 + 2 * RADIUS));
    extern __shared__ char sh_lattice[];

    size_t global_id;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    int blockStart_x = blockIdx.x * blockDim.x * 2;
    int blockStart_y = blockIdx.y * blockDim.y;

    for (int sh_id = tid; sh_id < totalShmem; sh_id += blockDim.x * blockDim.y)
    {
        int shmem_y = sh_id / (sh_stride);
        int shmem_x = sh_id % (sh_stride);

        global_id = (blockStart_y + shmem_y) * (size_t)(size_i + halo) + blockStart_x + shmem_x;
        if (blockStart_y + shmem_y < size_i + halo && blockStart_x + shmem_x < size_j + halo)
        {
            sh_lattice[sh_id] = d_lattice[global_id];
        }
    }
    // __syncthreads();
    // if (blockIdx.x + blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
    //     printf("SHMEM: %d\n", totalShmem);
    //     for (size_t sh_id = 0 ; sh_id < totalShmem; sh_id+=1) {
    //         printf("%d ", sh_lattice[sh_id]);
    //         if (sh_id % (BSIZE3DX*2 + 2*RADIUS) == BSIZE3DX*2 + 2*RADIUS - 1) printf("\n");
    //     }

    // }

    __syncthreads();

    uint32_t subcell[2] = {0, 0};

    // col izq
    for (int ry = -RADIUS; ry <= RADIUS; ry++)
    {
        int y = threadIdx.y + ry + RADIUS;
        int x = threadIdx.x * 2;
        int sh_id = y * sh_stride + x;
        // if (y < BSIZE3DY + 2*RADIUS && x < BSIZE3DX*2 + 2*RADIUS){
        int c = sh_lattice[sh_id];
        subcell[0] += c;
        //}
    }

    // centro comun
    for (int ry = -RADIUS; ry <= RADIUS; ry++)
    {
        for (int rx = -RADIUS + 1; rx <= RADIUS; rx++)
        {
            int y = threadIdx.y + ry + RADIUS;
            int x = threadIdx.x * 2 + rx + RADIUS;
            int sh_id = y * sh_stride + x;
            // if (y < BSIZE3DY + 2*RADIUS && x < BSIZE3DX*2 + 2*RADIUS){
            int c = sh_lattice[sh_id];
            subcell[0] += c;
            subcell[1] += c;
            //}
        }
    }

    // col der
    for (int ry = -RADIUS; ry <= RADIUS; ry++)
    {
        int y = threadIdx.y + ry + RADIUS;
        int x = threadIdx.x * 2 + 2 * RADIUS + 1;
        int sh_id = y * sh_stride + x;
        // if (y < BSIZE3DY + 2*RADIUS && x < BSIZE3DX*2 + 2*RADIUS){
        int c = sh_lattice[sh_id];
        subcell[1] += c;
        //}
    }

    int global_x = blockStart_x + threadIdx.x * 2 + RADIUS;
    int global_y = blockStart_y + threadIdx.y + RADIUS;

    if (global_x < size_j + RADIUS && global_y < size_i + RADIUS)
    {
        size_t my_id = global_y * (size_t)(size_i + halo) + global_x;
        int c = sh_lattice[(threadIdx.y + RADIUS) * sh_stride + (threadIdx.x * 2 + RADIUS)];
        int c2 = sh_lattice[(threadIdx.y + RADIUS) * sh_stride + (threadIdx.x * 2 + RADIUS + 1)];
        subcell[0] -= c;
        subcell[1] -= c2;
        d_lattice_new[my_id] = c * h(subcell[0], SMIN, SMAX) + (1 - c) * h(subcell[0], BMIN, BMAX);
        d_lattice_new[my_id + 1] = c2 * h(subcell[1], SMIN, SMAX) + (1 - c2) * h(subcell[1], BMIN, BMAX);
        // d_lattice_new[my_id] = subcell[0];
        // d_lattice_new[my_id+1] = subcell[1];
    }
}

#define NEIGHS1
__forceinline__ __device__ int count_neighs(char c, int my_id, int size_i, char *lattice, int neighs, int halo)
{
    size_t size = size_i + halo;
    int count = 0;

#if RADIUS > 5
    for (int i = -RADIUS; i <= RADIUS; i++)
    {
#pragma unroll
        for (int j = -RADIUS; j <= RADIUS; j++)
        {
            count += lattice[my_id + i * size + j];
        }
    }
    count -= c;
    return count;
#endif
#if RADIUS > 0
    count = lattice[my_id - size - 1];
    count += lattice[my_id - size];
    count += lattice[my_id - size + 1];
    count += lattice[my_id - 1];
    count += lattice[my_id + 1];
    count += lattice[my_id + size - 1];
    count += lattice[my_id + size];
    count += lattice[my_id + size + 1];
#endif

#if RADIUS > 1
    int size2 = 2 * size;

    count += lattice[my_id - size2 - 2];
    count += lattice[my_id - size2 - 1];
    count += lattice[my_id - size2];
    count += lattice[my_id - size2 + 1];
    count += lattice[my_id - size2 + 2];

    count += lattice[my_id - size - 2];
    count += lattice[my_id - size + 2];

    count += lattice[my_id - 2];
    count += lattice[my_id + 2];

    count += lattice[my_id + size - 2];
    count += lattice[my_id + size + 2];

    count += lattice[my_id + size2 - 2];
    count += lattice[my_id + size2 - 1];
    count += lattice[my_id + size2];
    count += lattice[my_id + size2 + 1];
    count += lattice[my_id + size2 + 2];
#endif

#if RADIUS > 2
    int size3 = 3 * size;
    count += lattice[my_id - size3 - 3];
    count += lattice[my_id - size3 - 2];
    count += lattice[my_id - size3 - 1];
    count += lattice[my_id - size3];
    count += lattice[my_id - size3 + 1];
    count += lattice[my_id - size3 + 2];
    count += lattice[my_id - size3 + 3];

    count += lattice[my_id - size2 - 3];
    count += lattice[my_id - size2 + 3];

    count += lattice[my_id - size - 3];
    count += lattice[my_id - size + 3];

    count += lattice[my_id - 3];
    count += lattice[my_id + 3];

    count += lattice[my_id + size - 3];
    count += lattice[my_id + size + 3];

    count += lattice[my_id + size2 - 3];
    count += lattice[my_id + size2 + 3];

    count += lattice[my_id + size3 - 3];
    count += lattice[my_id + size3 - 2];
    count += lattice[my_id + size3 - 1];
    count += lattice[my_id + size3];
    count += lattice[my_id + size3 + 1];
    count += lattice[my_id + size3 + 2];
    count += lattice[my_id + size3 + 3];
#endif

#if RADIUS > 3
    int size4 = 4 * size;

    count += lattice[my_id - size4 - 4];
    count += lattice[my_id - size4 - 3];
    count += lattice[my_id - size4 - 2];
    count += lattice[my_id - size4 - 1];
    count += lattice[my_id - size4];
    count += lattice[my_id - size4 + 1];
    count += lattice[my_id - size4 + 2];
    count += lattice[my_id - size4 + 3];
    count += lattice[my_id - size4 + 4];

    count += lattice[my_id - size3 - 4];
    count += lattice[my_id - size3 + 4];

    count += lattice[my_id - size2 - 4];
    count += lattice[my_id - size2 + 4];

    count += lattice[my_id - size - 4];
    count += lattice[my_id - size + 4];

    count += lattice[my_id - 4];
    count += lattice[my_id + 4];

    count += lattice[my_id + size - 4];
    count += lattice[my_id + size + 4];

    count += lattice[my_id + size2 - 4];
    count += lattice[my_id + size2 + 4];

    count += lattice[my_id + size3 - 4];
    count += lattice[my_id + size3 + 4];

    count += lattice[my_id + size4 - 4];
    count += lattice[my_id + size4 - 3];
    count += lattice[my_id + size4 - 2];
    count += lattice[my_id + size4 - 1];
    count += lattice[my_id + size4];
    count += lattice[my_id + size4 + 1];
    count += lattice[my_id + size4 + 2];
    count += lattice[my_id + size4 + 3];
    count += lattice[my_id + size4 + 4];
#endif

#if RADIUS > 4
    int size5 = 5 * size;

    count += lattice[my_id - size5 - 5];
    count += lattice[my_id - size5 - 4];
    count += lattice[my_id - size5 - 3];
    count += lattice[my_id - size5 - 2];
    count += lattice[my_id - size5 - 1];
    count += lattice[my_id - size5];
    count += lattice[my_id - size5 + 1];
    count += lattice[my_id - size5 + 2];
    count += lattice[my_id - size5 + 3];
    count += lattice[my_id - size5 + 4];
    count += lattice[my_id - size5 + 5];

    count += lattice[my_id - size4 - 5];
    count += lattice[my_id - size4 + 5];

    count += lattice[my_id - size3 - 5];
    count += lattice[my_id - size3 + 5];

    count += lattice[my_id - size2 - 5];
    count += lattice[my_id - size2 + 5];

    count += lattice[my_id - size - 5];
    count += lattice[my_id - size + 5];

    count += lattice[my_id - 5];
    count += lattice[my_id + 5];

    count += lattice[my_id + size - 5];
    count += lattice[my_id + size + 5];

    count += lattice[my_id + size2 - 5];
    count += lattice[my_id + size2 + 5];

    count += lattice[my_id + size3 - 5];
    count += lattice[my_id + size3 + 5];

    count += lattice[my_id + size4 - 5];
    count += lattice[my_id + size4 + 5];

    count += lattice[my_id + size5 - 5];
    count += lattice[my_id + size5 - 4];
    count += lattice[my_id + size5 - 3];
    count += lattice[my_id + size5 - 2];
    count += lattice[my_id + size5 - 1];
    count += lattice[my_id + size5];
    count += lattice[my_id + size5 + 1];
    count += lattice[my_id + size5 + 2];
    count += lattice[my_id + size5 + 3];
    count += lattice[my_id + size5 + 4];
    count += lattice[my_id + size5 + 5];
#endif

    return count;
}

__global__ void copy_Rows(int size_i, char *d_lattice, int neighs, int halo)
{
    size_t my_id = (size_t)blockDim.x * blockIdx.x + threadIdx.x + neighs;
    int i = 0;
    size_t size = size_i + halo;

    if (my_id < (size_i + neighs))
    {
        for (i = 0; i < neighs; i++)
        {
            d_lattice[size * (size_i + (i + neighs)) + my_id] =
                d_lattice[(i + neighs) * size + my_id];                           // copia primeras filas en ultimas
            d_lattice[i * size + my_id] = d_lattice[size * (size_i + i) + my_id]; // copia ultimas filas en primeras
        }
    }
}

__global__ void copy_Cols(int size_i, char *d_lattice, int neighs, int halo)
{
    size_t my_id = (size_t)blockDim.x * blockIdx.x + threadIdx.x;
    int i = 0;
    // Al haber copiado la primer fila en la ultima columna, se puede directamente copiar la primer columna completa,
    // incluidas las ghost cells, en la ultima columna ghost, y las esquinas van a tener el valor apropiado, la esquina
    // diagonal opuesta.
    size_t size = size_i + halo;

    if (my_id < size)
    {
        for (i = 0; i < neighs; i++)
        {
            d_lattice[my_id * size + (size_i + (i + neighs))] =
                d_lattice[my_id * size + (i + neighs)];                           // copia primeras columnas en ultimas
            d_lattice[my_id * size + i] = d_lattice[my_id * size + (size_i + i)]; // copia ultimas columnas en primeras
        }
    }
}

#define my_id_topa ((size_t)y * (size_i + halo) + x)
#define col_topa (threadIdx.x + neighs)
#define row_topa (threadIdx.y + neighs)
#define my_sh_id_topa ((size_t)(row_topa) * (blockDim.x + halo) + (col_topa))
#define row_topa2 (warpId + neighs)

__global__ void SHARED_KERNEL(char *d_lattice, char *d_lattice_new, int size_i, int size_j, int neighs, int halo)
{
    int warpId = (threadIdx.y * blockDim.x + threadIdx.x) / 32;

    int count = 0;
    size_t x = blockDim.x * blockIdx.x + threadIdx.x + neighs;
    size_t y = blockDim.y * blockIdx.y + threadIdx.y + neighs;
    int v = 0;

    extern __shared__ char sh_lattice[];

    // interior
    if (y < size_i + neighs && x < size_j + neighs)
    {
        sh_lattice[my_sh_id_topa] = d_lattice[my_id_topa];
    }
    // halo
    size_t y2 = blockDim.y * blockIdx.y + warpId + neighs;
    // y= blockDim.y * blockIdx.y + warpId + neighs;
    if (warpId == 0 || warpId == 1)
    {
        for (v = 0; v < neighs; v++)
        {
            int gy = y2 - ((row_topa2)-neighs);
            size_t up_or_down = ((blockDim.x + neighs) * ((row_topa2)-neighs)) + v;

            sh_lattice[(up_or_down) * (blockDim.x + halo) + col_topa] =
                d_lattice[(gy - neighs + up_or_down) * (size_i + halo) + x];
            // printf("row=%d v=%d -- (%d,%d)-> (%d,%d)=%d\n",row, v, row,col,   up_or_down,col, d_lattice[(gy - neighs
            // + (up_or_down)) * (size_i + halo) + x]);

            // Corner Halos: left-up and left-down
            if ((col_topa - neighs) < neighs)
            {
                sh_lattice[(up_or_down) * (blockDim.x + halo) + (col_topa - neighs)] =
                    d_lattice[(gy - neighs + up_or_down) * (size_i + halo) + (x - neighs)];
                // printf("row=%d v=%d -- (%d,%d)-> (%d,%d)=%d\n",row, v, row, col,  up_or_down, col-neighs,
                // d_lattice[(gy - neighs + (up_or_down)) * (size_i + halo) + (x-neighs)]);
            }

            // Corner Halos: right-up and right-down
            if ((col_topa + neighs) >= blockDim.y + neighs)
            {
                sh_lattice[(up_or_down) * (blockDim.x + halo) + (col_topa + neighs)] =
                    d_lattice[(gy - neighs + up_or_down) * (size_i + halo) + (x + neighs)];
                // printf("row=%d v=%d -- (%d,%d)-> (%d,%d)=%d\n",row, v, row, col,  up_or_down, col+neighs,
                // sh_lattice[(up_or_down) * (blockDim.x+halo) + (col+neighs)] );
            }
        }
    }
    else if (warpId == 2 || warpId == 3)
    {
        for (v = 0; v < neighs; v++)
        {
            int gy = y2 - ((row_topa2)-neighs);
            int gx = x - (col_topa - neighs);
            int lr = ((blockDim.y + neighs) * ((row_topa2) & 1)) + v;

            // printf("row=%d v=%d -- (%d,%d)-> (%d,%d)=%d\n",row, v, col, row,  col, lr, d_lattice[(gx - neighs + lr) +
            // (gy + (col-neighs)) * (size_i + halo)]);
            sh_lattice[col_topa * (blockDim.x + halo) + lr] =
                d_lattice[(gx - neighs + lr) + (gy + (col_topa - neighs)) * (size_i + halo)];
        }
    }

    __syncthreads();

    if (x < size_i + neighs && y < size_j + neighs)
    {
        // if (i <= size_i && j <= size_j && (ii-1) != 0 && (ii-1) != blockDim.x && (jj-1) != 0 && (jj-1) != blockDim.y)
        // {
        char c = sh_lattice[my_sh_id_topa];

        count = count_neighs(
            c, my_sh_id_topa, blockDim.x, sh_lattice, neighs,
            halo); // decrease sh_size_x by 2 to use the same count_neighs function than the rest of the implementations
        d_lattice_new[my_id_topa] = c * h(count, SMIN, SMAX) + (1 - c) * h(count, BMIN, BMAX);
    }
}
//////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void kernel_init_lookup_table(int *GPU_lookup_table)
{
    int(*lookup_table)[CAGIGAS_CELL_NEIGHBOURS + 1] = (int(*)[CAGIGAS_CELL_NEIGHBOURS + 1]) GPU_lookup_table;

    if (threadIdx.y < 2 && blockIdx.x < (CAGIGAS_CELL_NEIGHBOURS + 1))
    {
        if (threadIdx.y == 0)
            if (blockIdx.x >= BMIN && blockIdx.x <= BMAX)
                lookup_table[threadIdx.y][blockIdx.x] = 1;
            else
                lookup_table[threadIdx.y][blockIdx.x] = 0;

        if (threadIdx.y == 1)
            if (blockIdx.x >= SMIN && blockIdx.x <= SMAX)
                lookup_table[threadIdx.y][blockIdx.x] = 1;
            else
                lookup_table[threadIdx.y][blockIdx.x] = 0;
    }
}

__global__ void ghostRows(uint64_t *grid, int ROW_SIZE, int GRID_SIZE, int horizontalHaloWidth, int verticalHaloSize)
{
    // We want id ∈ [1,GRID_SIZE]
    size_t my_id = blockDim.x * blockIdx.x + threadIdx.x + horizontalHaloWidth;
    int fullHorizontalSize = ROW_SIZE + 2 * horizontalHaloWidth;

    if (my_id < (ROW_SIZE + horizontalHaloWidth))
    {
        for (int currentHalo = 0; currentHalo < verticalHaloSize; currentHalo++)
        {
            // fill bottom halo
            grid[(currentHalo + verticalHaloSize + GRID_SIZE) * fullHorizontalSize + my_id] =
                grid[(currentHalo + verticalHaloSize) * fullHorizontalSize + my_id];

            // fill top halo
            grid[currentHalo * fullHorizontalSize + my_id] =
                grid[(currentHalo + GRID_SIZE) * fullHorizontalSize + my_id];
        }
    }
}

__global__ void ghostCols(uint64_t *grid, int ROW_SIZE, int GRID_SIZE, int horizontalHaloWidth, int verticalHaloSize)
{
    // We want id ∈ [0,SIZE+1]
    size_t my_id = blockDim.x * blockIdx.x + threadIdx.x;
    int fullHorizontalSize = ROW_SIZE + 2 * horizontalHaloWidth;
    int fullVerticalSize = GRID_SIZE + 2 * verticalHaloSize;

    if (my_id < fullVerticalSize)
    {
        for (int currentHalo = 0; currentHalo < horizontalHaloWidth; currentHalo++)
        {
            // Copy first real column to right most ghost column
            grid[(my_id) * (fullHorizontalSize) + horizontalHaloWidth + currentHalo + ROW_SIZE] =
                grid[(my_id) * (fullHorizontalSize) + horizontalHaloWidth + currentHalo];
            // Copy last real column to left most ghost column
            grid[my_id * (fullHorizontalSize) + currentHalo] =
                grid[my_id * (fullHorizontalSize) + currentHalo + ROW_SIZE];
        }
    }
}

__device__ inline int dist(int x0, int x1)
{
    return abs(x0 - x1);
}

// Original CAGIGAS code for r=1
__global__ void CAGIGAS_KERNEL(uint64_t *grid, uint64_t *newGrid, int *GPU_lookup_table, int ROW_SIZE, int GRID_SIZE)
{
    // We want id ∈ [1,SIZE]
    int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
    int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int id = iy * (ROW_SIZE + 2) + ix;
    uint64_t cell, new_cell = 0;
    uint64_t up_cell, down_cell, right_cell, left_cell;                // Up,down,right,left cells
    uint64_t upleft_cell, downleft_cell, upright_cell, downright_cell; // Diagonal cells
    unsigned char subcell;

    int k, numNeighbors;
    int(*lookup_table)[CELL_NEIGHBOURS + 1] = (int(*)[CELL_NEIGHBOURS + 1]) GPU_lookup_table;

    if (iy > 0 && iy <= GRID_SIZE && ix > 0 && ix <= ROW_SIZE)
    {
        cell = grid[id];

        // First (0) subcell:
        up_cell = grid[id - (ROW_SIZE + 2)];
        down_cell = grid[id + (ROW_SIZE + 2)];
        left_cell = grid[id - 1];
        upleft_cell = grid[id - (ROW_SIZE + 3)];
        downleft_cell = grid[id + (ROW_SIZE + 1)];

        numNeighbors = getSubCellD(up_cell, 0) + getSubCellD(down_cell, 0);                   // upper lower
        numNeighbors += getSubCellD(left_cell, ELEMENTS_PER_CELL - 1) + getSubCellD(cell, 1); // left right
        numNeighbors += getSubCellD(upleft_cell, ELEMENTS_PER_CELL - 1) +
                        getSubCellD(downleft_cell, ELEMENTS_PER_CELL - 1);   // diagonals left
        numNeighbors += getSubCellD(up_cell, 1) + getSubCellD(down_cell, 1); // diagonals right
        subcell = getSubCellD(cell, 0);
        setSubCellD(&new_cell, 0, lookup_table[subcell][numNeighbors]);

        // Middle subcells:
        for (k = 1; k < CELL_NEIGHBOURS - 1; k++)
        {
            numNeighbors = getSubCellD(up_cell, k) + getSubCellD(down_cell, k);          // upper lower
            numNeighbors += getSubCellD(cell, k - 1) + getSubCellD(cell, k + 1);         // left right
            numNeighbors += getSubCellD(up_cell, k - 1) + getSubCellD(down_cell, k - 1); // diagonals left
            numNeighbors += getSubCellD(up_cell, k + 1) + getSubCellD(down_cell, k + 1); // diagonals right
            subcell = getSubCellD(cell, k);
            setSubCellD(&new_cell, k, lookup_table[subcell][numNeighbors]);
        }

        // Last (CELL_NEIGHBOURS-1) subcell:
        right_cell = grid[id + 1];
        upright_cell = grid[id - (ROW_SIZE + 1)];
        downright_cell = grid[id + (ROW_SIZE + 3)];

        numNeighbors =
            getSubCellD(up_cell, ELEMENTS_PER_CELL - 1) + getSubCellD(down_cell, ELEMENTS_PER_CELL - 1); // upper lower
        numNeighbors += getSubCellD(cell, ELEMENTS_PER_CELL - 2) + getSubCellD(right_cell, 0);           // left right
        numNeighbors += getSubCellD(up_cell, ELEMENTS_PER_CELL - 2) +
                        getSubCellD(down_cell, ELEMENTS_PER_CELL - 2);                 // diagonals left
        numNeighbors += getSubCellD(upright_cell, 0) + getSubCellD(downright_cell, 0); // diagonals right
        subcell = getSubCellD(cell, ELEMENTS_PER_CELL - 1);
        setSubCellD(&new_cell, ELEMENTS_PER_CELL - 1, lookup_table[subcell][numNeighbors]);

        // Copy new_cell to newGrid:
        newGrid[id] = new_cell;
    }
}

__global__ void PACK_KERNEL(uint64_t *grid, uint64_t *newGrid, int *GPU_lookup_table, int ROW_SIZE, int GRID_SIZE,
                            int horizontalHaloWidth, int verticalHaloSize)
{
    // We want id ∈ [1,SIZE]
    int iy = blockDim.y * blockIdx.y + threadIdx.y + verticalHaloSize;
    int ix = blockDim.x * blockIdx.x + threadIdx.x + horizontalHaloWidth;
    int fullHorizontalSize = ROW_SIZE + 2 * horizontalHaloWidth;
    int fullVerticalSize = GRID_SIZE + 2 * verticalHaloSize;
    size_t fullSharedWidth = blockDim.x + 2 * horizontalHaloWidth;
    size_t id = iy * (fullHorizontalSize) + ix;
    int current_cell_idx = threadIdx.x + horizontalHaloWidth;
    int current_cell_idy = threadIdx.y + verticalHaloSize;
    size_t sh_id = (current_cell_idy) * (fullSharedWidth) + current_cell_idx;
    uint64_t center_cell, new_cell = 0;
    unsigned char subcell;
    int(*lookup_table)[CAGIGAS_CELL_NEIGHBOURS + 1] = (int(*)[CAGIGAS_CELL_NEIGHBOURS + 1]) GPU_lookup_table;
    extern __shared__ uint64_t sh_grid[];
    int blockStart_x = blockIdx.x * blockDim.x;
    int blockStart_y = blockIdx.y * blockDim.y;

    for (int i = threadIdx.y; i < BSIZE3DY + 2 * verticalHaloSize; i += BSIZE3DY)
    {
        for (int j = threadIdx.x; j < BSIZE3DX + 2 * horizontalHaloWidth; j += BSIZE3DX)
        {
            if ((blockStart_y + i) < fullVerticalSize && blockStart_x + j < fullHorizontalSize)
            {
                sh_grid[i * (BSIZE3DX + 2 * horizontalHaloWidth) + j] =
                    grid[(blockStart_y + i) * fullHorizontalSize + blockStart_x + j];
            }
        }
    }
    __syncthreads();

    uint32_t subcells[ELEMENTS_PER_CELL] = {0, 0, 0, 0, 0, 0, 0, 0};
    // unsigned char subcells[ELEMENTS_PER_CELL] = {0,0,0,0,0,0,0,0};
    uint64_t threadWord = sh_grid[sh_id];
    uint64_t left[2] = {0, 0};
    uint64_t right[2] = {0, 0};
    if (iy >= verticalHaloSize && iy < GRID_SIZE + verticalHaloSize && ix >= horizontalHaloWidth &&
        ix < ROW_SIZE + horizontalHaloWidth)
    {
        for (int i = -RADIUS; i <= RADIUS; i++)
        {
            int currentNeighPosition_y = threadIdx.y + verticalHaloSize + i;
            int currentNeighPosition_x = (threadIdx.x + horizontalHaloWidth);
            int currentNeighCellIndex = currentNeighPosition_y * fullSharedWidth + currentNeighPosition_x;
            // read the corresponding 64-bit words from x-neighborhood, once per i-row
            uint64_t centerWord = sh_grid[currentNeighCellIndex];
            left[0] = sh_grid[currentNeighCellIndex - 1];
            right[0] = sh_grid[currentNeighCellIndex + 1];
#if RADIUS > 8
            left[1] = sh_grid[currentNeighCellIndex - 2];
            right[1] = sh_grid[currentNeighCellIndex + 2];
#endif

// LEFT LOOP
#pragma unroll
            for (int j = -RADIUS; j < 0; j++)
            {
                int currentNeighSubcellIndex = (j) & (ELEMENTS_PER_CELL - 1);
                uint64_t currentNeighCell = left[((-j) - 1) >> 3];
                unsigned char subcell = getSubCellD(currentNeighCell, currentNeighSubcellIndex);
                int from = max(0, j - RADIUS);
                int to = min(7, j + RADIUS);
                for (int k = from; k <= to; k++)
                {
                    subcells[k] += subcell;
                }
            }

// CENTER LOOP
#pragma unroll
            for (int j = 0; j < 8; j++)
            {
                unsigned char subcell = getSubCellD(centerWord, j);
                if (i == 0)
                {
                    int from = max(0, j - RADIUS);
                    int to = j - 1;
                    for (int k = from; k <= to; k++)
                    {
                        subcells[k] += subcell;
                    }
                    from = j + 1;
                    to = min(7, j + RADIUS);
                    for (int k = from; k <= to; k++)
                    {
                        subcells[k] += subcell;
                    }
                }
                else
                {
                    int from = max(0, j - RADIUS);
                    int to = min(7, j + RADIUS);
                    for (int k = from; k <= to; k++)
                    {
                        subcells[k] += subcell;
                    }
                }
            }
// RIGHT LOOP
#pragma unroll
            for (int j = 8; j < 8 + RADIUS; j++)
            {
                int currentNeighSubcellIndex = (j) & (ELEMENTS_PER_CELL - 1);
                uint64_t currentNeighCell = right[(j - 8) >> 3];
                unsigned char subcell = getSubCellD(currentNeighCell, currentNeighSubcellIndex);
                int from = max(0, j - RADIUS);
                int to = min(7, j + RADIUS);
                for (int k = from; k <= to; k++)
                {
                    subcells[k] += subcell;
                }
            }
        }
// TRANSITION STATES
#pragma unroll
        for (int i = 0; i < ELEMENTS_PER_CELL; i++)
        {
            setSubCellD(&new_cell, i, lookup_table[getSubCellD(threadWord, i)][subcells[i]]);
        }
        // WRITE NEW 64-bit WORD
        newGrid[id] = new_cell;
    }
}

__forceinline__ unsigned char getSubCellH(uint64_t cell, char pos)
{
    return (cell >> (ELEMENTS_PER_CELL - 1 - pos) * 8);
}

__forceinline__ void setSubCellH(uint64_t *cell, char pos, unsigned char subcell)
{
    uint64_t mask = 0xFF;
    uint64_t maskNewCell = subcell;

    // Erase pos content in cell:
    mask = mask << (ELEMENTS_PER_CELL - 1 - pos) * 8;
    mask = ~mask;
    *cell = *cell & mask;

    // Add subcell content to cell in pos:
    *cell = *cell | (maskNewCell << (ELEMENTS_PER_CELL - 1 - pos) * 8);
}

__device__ unsigned char getSubCellD(uint64_t cell, char pos)
{
    return (cell >> (ELEMENTS_PER_CELL - 1 - pos) * 8);
}

__device__ void setSubCellD(uint64_t *cell, char pos, unsigned char subcell)
{
    uint64_t mask = 0xFF;
    uint64_t maskNewCell = subcell;

    // Erase pos content in cell:
    mask = mask << (ELEMENTS_PER_CELL - 1 - pos) * 8;
    mask = ~mask;
    *cell = *cell & mask;

    // Add subcell content to cell in pos:
    *cell = *cell | (maskNewCell << (ELEMENTS_PER_CELL - 1 - pos) * 8);
}

__global__ void unpackStateKernel(uint64_t *from, int *to, int ROW_SIZE, int GRID_SIZE, int horizontalHaloWidth,
                                  int verticalHaloSize)
{
    // We want id ∈ [1,SIZE]
    size_t unpacked_x = (blockDim.x * blockIdx.x + threadIdx.x) * 8 + verticalHaloSize;
    size_t unpacked_y = blockDim.y * blockIdx.y + threadIdx.y + verticalHaloSize;

    size_t packed_x = (blockDim.x * blockIdx.x + threadIdx.x) + horizontalHaloWidth;

    size_t unpackedIndex = unpacked_y * (GRID_SIZE + 2 * verticalHaloSize) + unpacked_x;
    size_t packedIndex = unpacked_y * (ROW_SIZE + 2 * horizontalHaloWidth) + packed_x;
    // print all i in one line

    uint64_t cellValue;
    unsigned char subcell;

    if (unpacked_y < GRID_SIZE + verticalHaloSize && unpacked_x < GRID_SIZE + verticalHaloSize)
    {
        cellValue = from[packedIndex];
        for (int i = 0; i < ELEMENTS_PER_CELL; i++)
        {
            subcell = getSubCellD(cellValue, i);
            to[unpackedIndex + i] = subcell;
        }
    }
}

__global__ void packStateKernel(int *from, uint64_t *to, int ROW_SIZE, int GRID_SIZE, int horizontalHaloWidth,
                                int verticalHaloSize)
{
    // We want id ∈ [1,SIZE]
    size_t unpacked_x = (blockDim.x * blockIdx.x + threadIdx.x) * 8 + verticalHaloSize;
    size_t unpacked_y = blockDim.y * blockIdx.y + threadIdx.y + verticalHaloSize;

    size_t packed_x = (blockDim.x * blockIdx.x + threadIdx.x) + horizontalHaloWidth;

    size_t unpackedIndex = unpacked_y * (GRID_SIZE + 2 * verticalHaloSize) + unpacked_x;
    size_t packedIndex = unpacked_y * (ROW_SIZE + 2 * horizontalHaloWidth) + packed_x;
    // print all i in one line
    uint64_t cellValue = 0;
    // if (threadIdx.x + threadIdx.y == 0) {
    //     for (int i = 0; i < GRID_SIZE + 2 * verticalHaloSize; i++) {
    //         for (int j = 0; j < GRID_SIZE + 2 * verticalHaloSize; j++) {
    //             printf("%d ", from[i * (GRID_SIZE + 2 * verticalHaloSize) + j]);
    //         }
    //         printf("\n");
    //     }
    // }

    // printf("(%d, %d) = %d\n", unpacked_x, unpacked_y, unpackedIndex);
    // printf("t(%i, %i) -> unpack(%i, %i) -> GRID_SIZE:%i,  verticalHalo:%i, total_width:%i\n", threadIdx.x,
    // threadIdx.y, unpacked_x, unpacked_y, GRID_SIZE, verticalHaloSize, GRID_SIZE + 2 * verticalHaloSize);
    if (unpacked_y < GRID_SIZE + verticalHaloSize && unpacked_x < GRID_SIZE + verticalHaloSize)
    {

        // printf("    t(%i, %i) -> unpack(%i, %i) -> GRID_SIZE:%i,  verticalHalo:%i, total_width:%i\n", threadIdx.x,
        // threadIdx.y, unpacked_x, unpacked_y, GRID_SIZE, verticalHaloSize, GRID_SIZE + 2 * verticalHaloSize);
        // printf("t(%i, %i) -> %d, %d -> %d %d %d %d %d %d %d %d\n",threadIdx.x, threadIdx.y, unpacked_x, unpacked_y,
        // from[unpackedIndex], from[unpackedIndex + 1], from[unpackedIndex + 2], from[unpackedIndex + 3],
        // from[unpackedIndex + 4], from[unpackedIndex + 5], from[unpackedIndex + 6], from[unpackedIndex + 7]);
        // printf("t(%i,%i) -> unpack(%llu,%llu) - GRID_SIZE: %i\n", threadIdx.x, threadIdx.y, unpacked_x, unpacked_y,
        // GRID_SIZE);

        for (int i = 0; i < ELEMENTS_PER_CELL; i++)
        {
            // unsigned char subcell = getSubCellD(cellValue, i);
            // printf("i,j = %llu, %llu = %i\n", unpacked_y, unpacked_x+i, subcell);
            setSubCellD(&cellValue, i, from[unpackedIndex + i]);
        }
        to[packedIndex] = cellValue;
        // printf("%d, %d -> cellValue=%lx\n", unpacked_y, unpacked_x, cellValue);
    }
}

// __global__ void GOL(uint64_t* grid, uint64_t* newGrid, int* GPU_lookup_table, int ROW_SIZE, int GRID_SIZE, int
// horizontalHaloWidth, int verticalHaloSize) {
//     // We want id ∈ [1,SIZE]
//     int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
//     int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
//     size_t id = iy * (ROW_SIZE + 2) + ix;
//     uint64_t cell, new_cell = 0;
//     uint64_t up_cell, down_cell, right_cell, left_cell;                 // Up,down,right,left cells
//     uint64_t upleft_cell, downleft_cell, upright_cell, downright_cell;  // Diagonal cells
//     unsigned char subcell;

//     int k, numNeighbors;
//     int(*lookup_table)[CAGIGAS_CELL_NEIGHBOURS + 1] = (int(*)[CAGIGAS_CELL_NEIGHBOURS + 1]) GPU_lookup_table;

//     if (iy > 0 && iy <= GRID_SIZE && ix > 0 && ix <= ROW_SIZE) {
//         cell = grid[id];

//         // First (0) subcell:
//         up_cell = grid[id - (ROW_SIZE + 2)];
//         down_cell = grid[id + (ROW_SIZE + 2)];
//         left_cell = grid[id - 1];
//         upleft_cell = grid[id - (ROW_SIZE + 3)];
//         downleft_cell = grid[id + (ROW_SIZE + 1)];

//         numNeighbors = getSubCellD(up_cell, 0) + getSubCellD(down_cell, 0); // upper lower numNeighbors +=
//         getSubCellD(left_cell, ELEMENTS_PER_CELL - 1) + getSubCellD(cell, 1);                                 // left
//         right numNeighbors += getSubCellD(upleft_cell, ELEMENTS_PER_CELL - 1) + getSubCellD(downleft_cell,
//         ELEMENTS_PER_CELL - 1);  // diagonals left numNeighbors += getSubCellD(up_cell, 1) + getSubCellD(down_cell,
//         1);                                                  // diagonals right subcell = getSubCellD(cell, 0);
//         setSubCellD(&new_cell, 0, lookup_table[subcell][numNeighbors]);

//         // Middle subcells:
//         for (k = 1; k < CAGIGAS_CELL_NEIGHBOURS - 1; k++) {
//             numNeighbors = getSubCellD(up_cell, k) + getSubCellD(down_cell, k);           // upper lower
//             numNeighbors += getSubCellD(cell, k - 1) + getSubCellD(cell, k + 1);          // left right
//             numNeighbors += getSubCellD(up_cell, k - 1) + getSubCellD(down_cell, k - 1);  // diagonals left
//             numNeighbors += getSubCellD(up_cell, k + 1) + getSubCellD(down_cell, k + 1);  // diagonals right
//             subcell = getSubCellD(cell, k);
//             setSubCellD(&new_cell, k, lookup_table[subcell][numNeighbors]);
//         }
//
//         // Last (CAGIGAS_CELL_NEIGHBOURS-1) subcell:
//         right_cell = grid[id + 1];
//         upright_cell = grid[id - (ROW_SIZE + 1)];
//         downright_cell = grid[id + (ROW_SIZE + 3)];

//         numNeighbors = getSubCellD(up_cell, ELEMENTS_PER_CELL - 1) + getSubCellD(down_cell, ELEMENTS_PER_CELL - 1);
//         // upper lower numNeighbors += getSubCellD(cell, ELEMENTS_PER_CELL - 2) + getSubCellD(right_cell, 0); // left
//         right numNeighbors += getSubCellD(up_cell, ELEMENTS_PER_CELL - 2) + getSubCellD(down_cell, ELEMENTS_PER_CELL
//         - 2);  // diagonals left numNeighbors += getSubCellD(upright_cell, 0) + getSubCellD(downright_cell, 0); //
//         diagonals right subcell = getSubCellD(cell, ELEMENTS_PER_CELL - 1); setSubCellD(&new_cell, ELEMENTS_PER_CELL
//         - 1, lookup_table[subcell][numNeighbors]);

//         // Copy new_cell to newGrid:
//         newGrid[id] = new_cell;

//         /*
//                 // Get the number of neighbors for a given grid point
//                 numNeighbors = grid[id+(SIZE+2)] + grid[id-(SIZE+2)] //upper lower
//                              + grid[id+1] + grid[id-1]             //right left
//                              + grid[id+(SIZE+3)] + grid[id-(SIZE+3)] //diagonals
//                              + grid[id-(SIZE+1)] + grid[id+(SIZE+1)];

//                 uint64_t cell = grid[id];
//                 newGrid[id] = lookup_table[cell][numNeighbors];
//         */
//     }
// }

#endif
